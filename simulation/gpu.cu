
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <iostream>

#define N 1048576

using uint = unsigned int;

// CUDA核函数，负责向量加法
__global__ void vectorAddition(const uint* A, const uint* B, uint* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

int main() {
    // 分配向量在主机内存上
    uint* h_A = new uint[N];
    uint* h_B = new uint[N];
    uint* h_C = new uint[N];

    // 初始化向量数据
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    // 分配GPU上的设备内存
    uint* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(uint));
    hipMalloc((void**)&d_B, N * sizeof(uint));
    hipMalloc((void**)&d_C, N * sizeof(uint));

    // 将向量数据从主机内存复制到设备内存
    hipMemcpy(d_A, h_A, N * sizeof(uint), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(uint), hipMemcpyHostToDevice);

    // 定义网格和块的大小
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;

    // 调用CUDA核函数进行向量加法
    for (int i = 0; i < 4000; ++i) {
        vectorAddition<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    }

    // 将结果从设备内存复制回主机内存
    hipMemcpy(h_C, d_C, N * sizeof(int), hipMemcpyDeviceToHost);

    // 释放内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
